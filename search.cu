#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""
#include <hip/device_functions.h>


__device__ int dev_result1[10000];
__device__ int dev_count[1] = { 0 };

__global__ void searchNumber(const float* a, int* result)
{
    int thread = blockIdx.x * blockDim.x + threadIdx.x;

    int i = thread;

    int shift = 0;
    while (shift < 2) {
        int j = i + shift * 32768;
        if (a[j] == 65500) {
            result[0] = j;
        }
        shift++;
    }
}

__global__ void charcheck(char* a)
{
    int thread = blockIdx.x * blockDim.x + threadIdx.x;

    int i = thread;

    a[i] = 'a';
}

__global__ void resetVars()
{
    memset(dev_result1, 0, 10000 * sizeof(*dev_result1));
    memset(dev_count, 0, 1 * sizeof(*dev_count));

}

__global__ void searchField(char* a, int fieldfieldsize, char* value,  int rowsize, int fieldsize, int totalrow,char* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //char val[9]= { ' ',' ','k', 'i', 'v', 'a', 'n', 'c','\0'};
    if (i < totalrow) {
        bool check = true;
        int start = i * 24 + 8;

        for (int j = 0; j < 8; j++) {
            if (value[j] != a[start]) {
                check = false;
                goto Exit;
            }
            start++;
        }

        if (check) {
            result[i] = 1;
        }
        Exit:

    }
}

class CudaTable {
public:             // Access specifier
    char* dev_rows;
    hipStream_t stream;
    char* dev_chararray;
    char* dev_resultarray;
    char resultarray[1048576] = { ' ' };

    CudaTable() {
        hipError_t cudaStatus;
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        }
        dev_rows = 0;
        hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
        
        
        hipMalloc((void**)&dev_chararray, 8 * sizeof(char));
        hipHostMalloc((void**)&resultarray, 1048576 * sizeof(char));
        hipMalloc((void**)&dev_resultarray, 1048576 * sizeof(char));
    }
    // Helper function for using CUDA to add vectors in parallel.
    int* searchWithCuda(const float* a, int* result, unsigned int size)
    {
        float* dev_a = 0;
        int* dev_result = 0;
        hipError_t cudaStatus;

        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }



        cudaStatus = hipMalloc((void**)&dev_result, 1 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_result, result, 1 * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        // Launch a kernel on the GPU with one thread for each element.
        searchNumber << <size / 256, 128 >> > (dev_a, dev_result);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Error;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(result, dev_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

    Error:
        hipFree(dev_a);
        hipFree(dev_result);

        return result;
    }

    void updateGPU(char* a, unsigned int size,int start) {
        hipError_t cudaStatus;
        cudaStatus = hipMemcpy(dev_rows+start, a, size * sizeof(char), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    Error:
        fprintf(stderr, "%s", cudaStatus);
    }
    void insertToGPU(char* a, unsigned int size) {

        hipError_t cudaStatus;

        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_rows, size * sizeof(char));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_rows, a, size * sizeof(char), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Error;
        }


    Error:
        fprintf(stderr, "hipMemcpy failed!");

    }
    

    // Helper function for using CUDA to add vectors in parallel.
    char* searchStringWithCudaGPU(unsigned int size, unsigned int totalsize, int field, char* value, int rowsize, int fieldsize)
    {
        
        std::vector<int> count(1);
        
        hipError_t cudaStatus;
        

        std::chrono::steady_clock::time_point begin_1 = std::chrono::steady_clock::now();

       /* cudaStatus = hipMalloc((void**)&dev_chararray, 8 * sizeof(char));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }*/

        
        std::chrono::steady_clock::time_point end_1 = std::chrono::steady_clock::now();
        std::cout << "Time difference malloc= " << std::chrono::duration_cast<std::chrono::microseconds>(end_1 - begin_1).count() << "[microseconds]" << std::endl;


       /* cudaStatus = hipMalloc((void**)&dev_result, 1 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }*/
        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
        //memcpy(dev_chararray, value, 8 * sizeof(char));

        cudaStatus = hipMemcpy(dev_chararray, value, 8 * sizeof(char), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
        
        cudaStatus = hipMemcpy(dev_resultarray, resultarray, 8 * sizeof(char), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[microseconds]" << std::endl;




        /*cudaStatus = hipMemcpy(dev_result, result, 1 * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }*/
        std::chrono::steady_clock::time_point begin0 = std::chrono::steady_clock::now();
        searchField << <1024, 1024 >> > (dev_rows, field*fieldsize, dev_chararray, rowsize, fieldsize, totalsize/rowsize,dev_resultarray);
        std::chrono::steady_clock::time_point end0 = std::chrono::steady_clock::now();
        std::cout << "Time difference0 = " << std::chrono::duration_cast<std::chrono::microseconds>(end0 - begin0).count() << "[microseconds]" << std::endl;

        // Launch a kernel on the GPU with one thread for each element.

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        std::chrono::steady_clock::time_point begin4 = std::chrono::steady_clock::now();
        
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Error;
        }
        std::chrono::steady_clock::time_point end4 = std::chrono::steady_clock::now();
        std::cout << "Time difference4 = " << std::chrono::duration_cast<std::chrono::microseconds>(end4 - begin4).count() << "[microseconds]" << std::endl;

        cudaStatus = hipMemcpy(resultarray, dev_resultarray, 1024*1024 * sizeof(char), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
        /*cudaStatus = hipMemcpyFromSymbol(&(count[0]), dev_count, 1 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!\n", cudaStatus);
            goto Error;
        }


        cudaStatus = hipMemcpyFromSymbol(&(result[0]), dev_result1, count[0] * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!\n", cudaStatus);
            goto Error;
        }*/
        

        //resetVars << <1,1 >> > ();
        
        

        // Copy output vector from GPU buffer to host memory.
        /*cudaStatus = hipMemcpy(result, dev_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }*/

    Error:
        
        return resultarray;
    }
};