#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""
#include <hip/device_functions.h>
#include "vector"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>
#include <algorithm>
#include "table.cpp"
#include "add.cu"

using namespace std;



int getSPcores(hipDeviceProp_t devProp);
#define SIZE 1024 // FOR PARALEL GPU  SIZE HAS TO BE 2^n 1024
#define THREADS 4 // FOR PARALEL GPU  THREADS = SIZE / (BLOCKS * 2) 
#define BLOCKS 128//FOR PARALEL GPU  BLOCKS = SIZE / (THREADS * 2) 
string type = "DEVICE";  // USE "HOST" FOR CPU BUBBLE SORT, USE "DEVICE" FOR GPU BUBBLE SORT
int flag = 0;

__device__ long d_answer = 0;




__global__ void sortArray(int* a, int* size)
{
    int thread = blockIdx.x * blockDim.x + threadIdx.x;

    int i = thread;
    
    if (size[0] % 2 == 0) {
        if (i < size[0] - 1) {
            if (i % 2 == 0) {
                if (a[i] < a[i + 1]) {
                    int temp = a[i];
                    a[i] = a[i + 1];
                    a[i + 1] = temp;
                }
            }
        }
    }


    if (size[0] % 2 == 0) {
        if (i % 2 != 0) {
            if (a[i] < a[i + 1]) {
                int temp = a[i];
                a[i] = a[i + 1];
                a[i + 1] = temp;
            }
        }
    }
    else {
        if (i < size[0] - 1 && i % 2 != 0) {
            if (a[i] < a[i + 1]) {
                int temp = a[i];
                a[i] = a[i + 1];
                a[i + 1] = temp;
            }
        }

    }
}


int main() {
    cout << "Hello World!\n";
    string command;
    vector<table*> tables;
    bool run = true;
    while (run) {
        cout << "Enter command: ";
        cin >> command;
        if (command == "create") {
            vector<string> fields(3);
            fields[0] = "id";
            fields[1] = "name";
            fields[2] = "surname";
            table* asd = new table("table1", fields);
            tables.push_back(asd);
        }

        if (command == "insert")
        {
            string flds;
            string vals;
            cout << "Enter fields comma seperated:";
            cin >> flds;
            cout << "Enter values comma seperated:";
            cin >> vals;
            tables[0]->insert(flds, vals);
        }

        if (command == "insert2")
        {

            tables[0]->insert50000();
        }

        if (command == "select") {
            vector<char> res = tables[0]->read();
            int size = res.size();
            for (size_t i = 0; i < size; i++)
            {
                cout << res[i];
            }
        }

        if (command == "search") {
            tables[0]->search("", "");
        }

        if (command == "sort") {
            tables[0]->sort();
        }

        if (command == "exit") {
            run = false;
        }

        if (command == "update")
        {
            string value;
            cout << "enter value:";
            cin >> value;
            tables[0]->update(value);
        }

    }
}

int main0()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    int result[1] = { -1 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize, result);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    printf("{%d}\n",
        result[0]);
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}







int getSPcores(hipDeviceProp_t devProp)
{
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
    case 2: // Fermi
        if (devProp.minor == 1) cores = mp * 48;
        else cores = mp * 32;
        break;
    case 3: // Kepler
        cores = mp * 192;
        break;
    case 5: // Maxwell
        cores = mp * 128;
        break;
    case 6: // Pascal
        if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
        else if (devProp.minor == 0) cores = mp * 64;
        else printf("Unknown device type\n");
        break;
    case 7: // Volta and Turing
        if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
        else printf("Unknown device type\n");
        break;
    case 8: // Ampere
        if (devProp.minor == 0) cores = mp * 64;
        else if (devProp.minor == 6) cores = mp * 128;
        else printf("Unknown device type\n");
        break;
    default:
        printf("Unknown device type\n");
        break;
    }
    return cores;
}